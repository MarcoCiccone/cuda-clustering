#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <time.h>
#include <hip/device_functions.h>
#include "cutil_inline.h"
#include <shrQATest.h>
#include <math.h>


    // OpenGL Graphics includes
#include <GL/glew.h>
#ifdef _WIN32
#include <GL/wglew.h>
#endif 
#if defined(__APPLE__) || defined(__MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

/*
 
 TODO:
 
 ripulire il codice opengl
 spostare il ricalcolo del centroide nel kernel cuda
 parametrizzare il numero dei kernel e il nome del file di testo.
 ricreare uno storico per ogni centroide.
 
 Spostare nelle funzioni dove � possibile.
 cercare lo zoom da mandelbrot
 
 ATTENZIONE per come � implementata adesso � inutile ricaricare i dati dei centroidi dal device
 
 h_odata
 */


/*
 Data on Device : 
 valpoint d_idata
 __constant__ centroid constData
 [d_centroids is not use anymore] now is used constData
 
 Data on Host : 
 valpoint h_idata
 centroid h_centroids
 
 */


/*
 
 STEPS OF THE ALGORITHM: 
 Step 1: Place randomly initial group centroids into the 2d space.
 Step 2: Assign each object to the group that has the closest centroid.
 Step 3: Recalculate the positions of the centroids.
 Step 4: If the positions of the centroids didn't change go to the next step, else go to Step 2.
 Step 5: End 
 */


    // Constants -----------------------------------------------------------------

#define kWindowWidth	1024
#define kWindowHeight	720
#define namefile "gps/firenze.gps"

#define XLucca 43.8
#define YLucca 10.5

#define XFirenze 43.7
#define YFirenze 11.2

#define offsetLongFirenze -1200.0
#define offsetLatFirenze  -300.0

#define offsetLongLucca -300.0
#define offsetLatLucca  300.0

typedef struct {
    float x;
    float y;
    int index_cluster;
}valpoint;

typedef struct {
    float x;
    float y;
    unsigned long numMembers;
        //centroid * history;
}centroid;


static valpoint* h_idata;
static centroid* h_centroids;
static unsigned long numElements;
static unsigned long numClusters = 68;
static const char* input_file;
static const char* output_file = "output.txt";

    // Function Prototypes -------------------------------------------------------

GLvoid InitGL(GLvoid);
GLvoid DrawGLScene(GLvoid);
GLvoid glCircle3f(GLfloat x, GLfloat y, GLfloat radius); 


    // InitGL -------------------------------------------------------------------

GLvoid InitGL(GLvoid)
{
    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);		// This Will Clear The Background Color To Black
        //Setup a 2D projection
    glMatrixMode (GL_PROJECTION);
    glLoadIdentity ();                          // Reset The Projection Matrix
    glOrtho (0, kWindowWidth, kWindowHeight, 0, 0, 1);
    glDisable(GL_DEPTH_TEST);
        // Calculate The Aspect Ratio Of The Window
    glMatrixMode (GL_MODELVIEW);
    glLoadIdentity();
    
    
}


    // DrawGLScene --------------------------------------------------------------

GLvoid DrawGLScene(GLvoid)
{    
	float XX=0,YY=0;
	float offsetLong=0 , offsetLat = 0;   
	if (strstr(input_file,"firenze") != NULL){
		XX = XFirenze;
		YY = YFirenze;
		offsetLong = offsetLongFirenze;
		offsetLat = offsetLatFirenze;
	}
	if (strstr(input_file,"lucca") != NULL){
		XX = XLucca;
		YY = YLucca;
		offsetLong = offsetLongLucca;
		offsetLat = offsetLatLucca;
	}
        //Displacement trick for exact pixelization
    glTranslatef(0.375, 0.375, 0);
        //Draw a scene
    glClear(GL_COLOR_BUFFER_BIT);
        //glColorPointer(3, GL_FLOAT, 0, colorArray);
    
    int i=0,Vertex=0;   
    GLfloat x1 = 0.0 , y1 = 0.0;
    
    
    offsetLong = kWindowWidth * offsetLong / 1440;
    offsetLat = kWindowHeight * offsetLat / 900;
    
    GLfloat colorArray [256 * 256][3]; // color array
    
        // loop over all vertices 
    srand (time(NULL)) ;
    for (int z = 0; z < 256; z++)
    {
        for (int x = 0; x < 256; x++)
        {
                // VERTEX - vertices are numbered left to right, top to bottom
            Vertex = (z * 256) + x;
            
                // COLOUR - set the values in the color array 
                // RGB] = set same colour value to all 3 colours
            colorArray[Vertex][0] = (rand()%255)/255.0;
            colorArray[Vertex][1] = (rand()%255)/255.0;
            colorArray[Vertex][2] = (rand()%255)/255.0;
                //printf("%f    %f    %f  \n",colorArray[Vertex][0],colorArray[Vertex][1],colorArray[Vertex][2]);
            
            
        }
    }
    
    for (i=0; i<numElements; i++) {    
        x1 = (h_idata[i].x-XX);
        y1 = (h_idata[i].y-YY);
        
            //26000 : 1440 = fattoreLong : kWindowWidth --->   fattoreLong = kWindowWidth * 26000 / 1440
        x1 = x1*(kWindowWidth * 26000 / 1440)+offsetLong;
        y1 = y1*(kWindowHeight * 13000 / 900)+offsetLat;
        
        /*if (x1 > kWindowWidth || y1>kWindowHeight){
         printf("Lat: %f --> px: %f \t Long: %f --> px: %f \n",h_idata[i].x,x1,h_idata[i].y,y1);
         
         getchar();
         }*/
        
        glBegin(GL_POINTS);// Start Drawing A Point
        glColor3f(colorArray[h_idata[i].index_cluster][0],colorArray[h_idata[i].index_cluster][1],colorArray[h_idata[i].index_cluster][2]);
        glVertex2f(x1, y1);
        glEnd();   
        
            //printf("Lat: %f --> px: %f \t Long: %f --> px: %f   index cluster : %i    \n",h_idata[i].x,x1,h_idata[i].y,y1,h_idata[i].index_cluster);
        
    }
    
        //disegno i centroidi
    for (i=0;i<numClusters;i++){
        x1 = (h_centroids[i].x-XX);
        y1 = (h_centroids[i].y-YY);
        
        x1 = x1*(kWindowWidth * 26000 / 1440)+offsetLong;
        y1 = y1*(kWindowHeight * 13000 / 900)+offsetLat;
        glColor3f(colorArray[i][0],colorArray[i][1],colorArray[i][2]);
        glCircle3f(x1,y1,5.0);
    }
    
    
        // When we've finished rendering the scene, we display it with
    glutSwapBuffers();
    
}

    // OpenGL keyboard function
void keyboardFunc(unsigned char k, int, int)
{
    switch (k){
        case '\033':
        case 'q':
        case 'Q':
            printf("Shutting down...\n");
            exit(EXIT_SUCCESS);
            break;
            
    }
}

__constant__ centroid constData[4096];
__global__ void KmeansKernel( valpoint* g_idata, centroid* g_centroids, int numClusters,int numElements) {
    
    unsigned long valindex = blockIdx.x * 512 + threadIdx.x ;
    
    if (valindex < numElements){
        int k, myCentroid;
        float minDistance;
        float euclideDistance;
        
        float X_sumCentroid = 0, Y_sumCentroid = 0;
        int countCentroid= 0;
        
        minDistance = 0xFFFFFFFF;
        
        for (k = 0; k<numClusters; k++){//per ogni valore della lista viene calcolata la distanza con il centroide di ogni cluster
                // calcolando le distanze controllo di quale cluster fa parte
            euclideDistance = (float)sqrt((pow(g_idata[valindex].x - g_centroids[k].x,2) + pow(g_idata[valindex].y - g_centroids[k].y,2)));
            if (euclideDistance < minDistance) {
                minDistance = euclideDistance;			
                myCentroid = k;
            }
        }
        
        g_idata[valindex].index_cluster = myCentroid;
            //g_centroids[myCentroid].numMembers++;
        
        __syncthreads();
        
        
            // quando hanno finito tutti i thread magari posso fare qualcosa che puo' essere paralizzata, come ad esempio il ricalcolo del centroide

        if (valindex < numClusters) {
            countCentroid= 0;
            X_sumCentroid=0;
            Y_sumCentroid=0;
                // faccio calcolare per ognuno di dei cluster il nuovo centroide
                //            printf("VALINDEX : %i \n\n",valindex);    
                //getchar();
            for (int i=0;i<numElements;i++){
                    //printf("value %f   %f  Cluster %i  \n",h_odata[i].x,h_odata[i].y,h_odata[i].index_cluster);
                if (g_idata[i].index_cluster == valindex){
                    X_sumCentroid += g_idata[i].x;
                    Y_sumCentroid += g_idata[i].y;
                    countCentroid++;
                }
            }
            if (countCentroid > 0){
                g_centroids[valindex].x = (float)(X_sumCentroid/countCentroid);
                g_centroids[valindex].y = (float)(Y_sumCentroid/countCentroid);
            }
            g_centroids[valindex].numMembers = countCentroid;
            
        }
        
        
            // atomicAdd(c,changes);
        __syncthreads();
        
        
        
    }
}



GLvoid glCircle3f(GLfloat x, GLfloat y, GLfloat radius) 
{ 
    float angle; 
    glLineWidth(1.0f); 
    glBegin(GL_LINE_LOOP); 
    for(int i = 0; i < 100; i++) { 
        angle = i*2*M_PI/100; 
        glVertex2f(x + (cos(angle) * radius), y + (sin(angle) * radius)); 
    } 
    glEnd(); 
}





/*INIZIO MAIN */
int main( int argc, char** argv) 
{
	if (argc != 3){
		printf("params : nameinputfile NumKluster");
		return 1;
	}

	numClusters = atoi(argv[2]);

	if (numClusters<1){
		printf("Il valore di k deve essere un intero maggiore di 1, verrà utilizzzato il default 68" );
		numClusters = 68;
	}
	
	input_file = argv[1];
	
    unsigned int mem_size;
    float gridDimension = 0;
    valpoint *d_idata, *h_odata;
    centroid *d_centroids;
    centroid* h_ocentroids;
    
    float changes =0,change;
    int i=0;
    
    timeval start;
    gettimeofday(&start, NULL);
    
    CUT_DEVICE_INIT(argc, argv);
        //CUT_SAFE_CALL( cutCreateTimer( &timer));
    
    
    
    /***************************************************/
        // initialize the memory reading from text file
    char *line = NULL;
    char linefix[120];
    FILE *inFilePtr;
    inFilePtr = fopen(input_file, "r+");
        
	if (inFilePtr == NULL) {
	   printf("Failed to open file %s",input_file);
	   return -1;
	}    
        //calculate number of points from text file
    while ( fgets ( linefix, sizeof linefix, inFilePtr ) != NULL ) /* read a line */{
        numElements++;
    }
    
        //    numClusters = numElements / 256;
    mem_size = numElements * sizeof(valpoint);
    
        // allocate host memory
    h_idata = (valpoint*) malloc( mem_size);
    h_centroids = (centroid*)malloc (numClusters * sizeof(centroid)); // questi sono quelli inziali e che poi sostituisco via via
    /*h_ocentroids = (centroid*)malloc (numClusters * sizeof(centroid));*/ // questi sono quelli che recupero dal calcolo sul device
    /***************************************************/
    
    
    
    /***************************************************/
    rewind(inFilePtr); // BOF
    for (i=0; i<numElements; i++) {
        
            //read latitude
        fscanf(inFilePtr, "%f", &h_idata[i].x);
            //read longitude
        fscanf(inFilePtr, "%f", &h_idata[i].y);
            //read image HTTP [NOT USED]
        fscanf(inFilePtr, "%s", &line);
            //printf("Lat: %f   Long %f  \n",x,y);
        
        h_idata[i].index_cluster = 999;
    }
    /***************************************************/
    
    
    
    /***************************************************/
    srand (time(NULL)) ;
    int randomIndex;
    for(i = 0; i < numClusters; ++i){
            //random choose the centroids
        randomIndex = rand() % numElements;
        h_centroids[i].x = h_idata[randomIndex].x;
        h_centroids[i].y = h_idata[randomIndex].y;
        h_centroids[i].numMembers = 0;
        
    }
    /***************************************************/
    
    /*for(i = 0; i < numClusters; ++i){
     printf("%f     %f  \n",h_centroids[i].x,h_centroids[i].y);
     }*/
    
        //CUT_SAFE_CALL( cutStartTimer( timer));
    int iteration=1;
    while(1){
        
        
        
            // allocate device memory for data points
        CUDA_SAFE_CALL(hipMalloc( (void**) &d_idata, mem_size));
            // copy data points to device  [src] d_idata -> [destination] h_idata
        CUDA_SAFE_CALL(hipMemcpy(d_idata,h_idata, mem_size, hipMemcpyHostToDevice) );
        
            // allocate device memory for data points
        CUDA_SAFE_CALL(hipMalloc( (void**) &d_centroids, sizeof(centroid)* numClusters));
        
            // copy centroids to device: [src symbol] h_centroids -> [destination device] constData (or d_centroids)
        CUDA_SAFE_CALL(hipMemcpy(d_centroids, h_centroids,sizeof(centroid)* numClusters,hipMemcpyHostToDevice));
        
            // setup execution parameters
        
            //printf("%i \n",numElements);
        
        
        /***************************************************/    
            //numero elenti:  54597 (COME SUDDIVIDERLI???) (faccio eseguire un po' piu di thread? e poi dentro non faccio niente se l'id del thread � maggiore del numero di elementi
        
        gridDimension = (float)numElements / 512; // per adesso la dimensione � fissata a 512 x 1 x 1 poi facciamo altre prove
        
        dim3 grid(ceil(gridDimension), 1); //2048 blocks.
            // numElements can be up to 32 Mega samples
        dim3 threads( 512, 1, 1); // each block having 512 threads. The maximum is 768
        
        /***************************************************/
        
        
            //printf("Main thread: about to dispatch kernel...\n");
        
        KmeansKernel<<< grid, threads >>>(d_idata, d_centroids, numClusters, numElements/*,&changes*/);
        
            // check if kernel execution generated and error
        CUT_CHECK_ERROR("Kernel execution failed");
            //allocate mem for the result on host side
        
        
        h_odata = (valpoint*) malloc( mem_size);
        h_ocentroids = (centroid*)malloc (numClusters * sizeof(centroid));
        
            // copy result from device to host
        CUDA_SAFE_CALL( hipMemcpy( h_odata,d_idata, mem_size, hipMemcpyDeviceToHost) );
        CUDA_SAFE_CALL( hipMemcpy( h_ocentroids,d_centroids, sizeof(centroid)* numClusters, hipMemcpyDeviceToHost) );
        
        
            //ricalcolo il centro di massa
            //questa parte adesso �  eseguita sul device per aumentare il livello di parallelismo.
        /***************************************************/
        /*changes = 0;
         valindex=0;
         while (valindex < numClusters) {
         countCentroid= 0;
         X_sumCentroid=0;
         Y_sumCentroid=0;
         // faccio calcolare per ognuno di dei cluster il nuovo centroide
         //            printf("VALINDEX : %i \n\n",valindex);    
         //getchar();
         for (int i=0;i<numElements;i++){
         //printf("value %f   %f  Cluster %i  \n",h_odata[i].x,h_odata[i].y,h_odata[i].index_cluster);
         if (h_odata[i].index_cluster == valindex){
         X_sumCentroid += h_odata[i].x;
         Y_sumCentroid += h_odata[i].y;
         countCentroid++;
         }
         }
         if (countCentroid > 0){
         
         //questo poi pu� essere sommato con atomicSUM CUDA
         h_ocentroids[valindex].x = (float)(X_sumCentroid/countCentroid);
         h_ocentroids[valindex].y = (float)(Y_sumCentroid/countCentroid);
         
         changes += (float)sqrt((pow(h_centroids[valindex].x - h_ocentroids[valindex].x,2) + pow(h_centroids[valindex].y - h_ocentroids[valindex].y,2)));
         
         }
         h_ocentroids[valindex].numMembers = countCentroid;
         
         valindex++;
         }
         /***************************************************/
        
        
            
        
         change = 0.0;
         changes = 0.0;
         for(i = 0; i < numClusters; ++i){
             change = (float)sqrt((pow(h_centroids[i].x - h_ocentroids[i].x,2) + pow(h_centroids[i].y - h_ocentroids[i].y,2)));
         
                 //printf("OLD : %f   %f  count %i -   NEW :  %f %f  Count: %i   Change : %f\n",h_centroids[i].x,h_centroids[i].y,h_centroids[i].numMembers,h_ocentroids[i].x,h_ocentroids[i].y,h_ocentroids[i].numMembers,change);
         
             changes+=change;
         }
            //printf("changes : %f \n",changes);
        
            //getchar();
        
        /*int ccc=0;
        for (i=0;i<numElements;i++){
            if (h_idata[i].index_cluster != h_odata[i].index_cluster){
                printf("Cluster 1    %i Cluster2    %i  \n",h_idata[i].index_cluster,h_odata[i].index_cluster);
                ccc++;
            }
            
        }
        printf("%i",ccc);
        getchar();
        */
        
        /*for (i=0;i<numElements;i++){
         printf("value %f   %f  Cluster %i  \n",h_idata[i].x,h_idata[i].y,h_idata[i].index_cluster);
         
         }
         h_idata = h_odata;
         
         getchar();
         for (i=0;i<numElements;i++){
         printf("value %f   %f  Cluster %i  \n",h_idata[i].x,h_idata[i].y,h_idata[i].index_cluster);
         }
         getchar();*/ 
        
        
            //copio dentro d_idata, d_centroids i nuovi d_odata, d_ocentroids 
        h_idata = h_odata;
        h_centroids = h_ocentroids;
        
        CUDA_SAFE_CALL(hipFree(d_idata));
        CUDA_SAFE_CALL(hipFree(d_centroids));
        if (changes <0.01)
            break;
        iteration++;
    }
    
    timeval end;
	gettimeofday(&end, NULL);
	double elapsed = end.tv_sec+end.tv_usec/1000000.0 - start.tv_sec-start.tv_usec/1000000.0;
	
	
	printf("Time elapsed : %f \n",elapsed);
    	
    printf("Iterations : %i \n",iteration);
    
    
    
    FILE *fp;
    fp=fopen("output.txt", "w");
    
    for (i=0;i<numElements;i++){
	 fprintf(fp, "value %f   %f  Cluster %i  \n",h_idata[i].x,h_idata[i].y,h_idata[i].index_cluster);
	 
	 }
    
    fclose(fp);
    
    
    printf("Starting GLUT main loop...\n");
    printf("\n");
    printf("Press [q] to exit\n");
    printf("\n");
    
        //inizia la parte che disegna i pixel con opengl
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(kWindowWidth, kWindowHeight); 
    glutInitWindowPosition (100, 100);
    glutCreateWindow (argv[0]);
    
    InitGL();
    
    glutDisplayFunc(DrawGLScene); 
    glutKeyboardFunc(keyboardFunc);
    
    glutMainLoop();
    
    
        //CUT_SAFE_CALL( cutStopTimer( timer));
        //    printf( "Time: \%f(ms)\n", cutGetTimerValue( timer));
        //    CUT_SAFE_CALL( cutDeleteTimer( timer));
    
    /*for (i=0;i<numElements;i++){
     printf("value %f   %f  Cluster %i  \n",h_odata[i].x,h_odata[i].y,h_odata[i].index_cluster);
     }*/
        // cleanup memory
    free( h_idata);
    free( h_odata);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUT_EXIT(argc, argv);
    
    /*FINE MAIN*/
    return 0;
}











